#include "hip/hip_runtime.h"

#include "cuTypes.h"

void cu::init(data_gpu* data, unsigned int h, unsigned int w, unsigned int c) {
    data->height = h;
    data->width = w;
    data->channels = c;
    switch (c) {
    case 1:
        data->channelDesc1UC = hipCreateChannelDesc<unsigned char>();
        data->channelDesc2UC = hipCreateChannelDesc<unsigned char>();
        break;
    case 3:
        data->channelDesc1UC = hipCreateChannelDesc<uchar3>();
        data->channelDesc2UC = hipCreateChannelDesc<uchar3>();
        break;
    case 4:
        data->channelDesc1UC = hipCreateChannelDesc<uchar4>();
        data->channelDesc2UC = hipCreateChannelDesc<uchar4>();
        break;
    }
    data->channelDescF = hipCreateChannelDesc<float>();

    hipMallocArray(&(data->left), &(data->channelDesc1UC), data->width, data->height);
    hipMallocArray(&(data->right), &(data->channelDesc2UC), data->width, data->height);
    hipMalloc(&(data->disparity), w * h * sizeof(float));
    hipMalloc(&(data->udisp), w * 64 * sizeof(uchar1));
}

int cu::update(data_gpu* data, unsigned char* left_cpu, unsigned char* right_cpu) {
    size_t size = (data->height) * (data->width) * (data->channels) * sizeof(unsigned char);
    hipMemcpyToArray(data->left, 0, 0, left_cpu, size, hipMemcpyHostToDevice);
    hipMemcpyToArray(data->right, 0, 0, right_cpu, size, hipMemcpyHostToDevice);
    return 1;
    // TODO: error handle
}

int cu::grab(data_gpu* data, float* out_cpu) {
    size_t size = (data->height) * (data->width) * sizeof(float);
    hipMemcpy(out_cpu, data->disparity, size, hipMemcpyDeviceToHost);
    return 1;
    // TODO: error handle
}
